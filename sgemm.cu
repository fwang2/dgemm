// nvcc sgemm.cu -lcublas -arch=sm_70 -o sgemm

#include <stdio.h>
#include <unistd.h>
#include <iostream>
#include <cstdlib>
#include <assert.h>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hipblas.h>

using namespace std;

const char *cublasGetErrorString(hipblasStatus_t status)
{
    switch (status)
    {
    case HIPBLAS_STATUS_SUCCESS:
        return "CUBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
        return "CUBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
        return "CUBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
        return "CUBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
        return "CUBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
        return "CUBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
        return "CUBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
        return "CUBLAS_STATUS_INTERNAL_ERROR";
    }
    return "unknown error";
}

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

inline hipblasStatus_t checkCublas(hipblasStatus_t result)
{
    if (result != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cublasGetErrorString(result));
        assert(result == HIPBLAS_STATUS_SUCCESS);
    }
    return result;
}

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on CPU
void CPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A)
{
    for (int i = 0; i < nr_rows_A * nr_cols_A; i++)
    {
        A[i] = (float)rand() / (float)(RAND_MAX);
    }
}
int main(int argc, char **argv)
{

    int min_mkn = 1024;
    int max_mkn = 4096*8;
    int repeats = 2;

    cout << "running with"
         << " min_mkn: [" << min_mkn << "]"
         << " max_mkn: [" << max_mkn << "]"
         << " repeats: " << repeats
         << endl;

    hipblasStatus_t stat;
    hipblasHandle_t handle;
    checkCublas(hipblasCreate(&handle));

    // Allocate 3 arrays on CPU

    float *h_A = (float *)malloc(max_mkn * max_mkn * sizeof(float));
    float *h_B = (float *)malloc(max_mkn * max_mkn * sizeof(float));
    float *h_C = (float *)malloc(max_mkn * max_mkn * sizeof(float));

    CPU_fill_rand(h_A, max_mkn, max_mkn);
    CPU_fill_rand(h_B, max_mkn, max_mkn);
    CPU_fill_rand(h_C, max_mkn, max_mkn);

    __half *d_A, *d_B, *d_C;
    checkCuda(hipMallocManaged((void **)&d_A, max_mkn * max_mkn * sizeof(__half)));
    checkCuda(hipMallocManaged((void **)&d_B, max_mkn * max_mkn * sizeof(__half)));
    checkCuda(hipMallocManaged((void **)&d_C, max_mkn * max_mkn * sizeof(float)));

    // TODO: just initialize on GPU
    for (int i = 0; i < max_mkn * max_mkn; i++)
    {
        d_A[i] = __float2half(h_A[i]);
        d_B[i] = __float2half(h_B[i]);
    }

    checkCuda(hipMemcpy(d_A, h_A, max_mkn * max_mkn * sizeof(__half), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_B, h_B, max_mkn * max_mkn * sizeof(__half), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_C, h_C, max_mkn * max_mkn * sizeof(float), hipMemcpyHostToDevice));

    cout << "Transferred A, B, C matrix from host to device" << endl;

    int lda, ldb, ldc, m, n, k;
    const float alf = 1.0f;
    const float bet = 0.0f;
    const float *alpha = &alf;
    const float *beta = &bet;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double totalSum = 0.0;
    for (int size = min_mkn; size <= max_mkn; size = size + 1024) {
        double sum = 0.0;        
        for(int rep = 0; rep < repeats; rep++) {
            hipEventRecord(start, 0);
            m = n = k = size;
            lda = m;
            ldb = k;
            ldc = m;
            // NT is the best
            stat = cublasSgemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, alpha,
                               (void *)d_A, HIP_R_16F, lda,
                               (void *)d_B, HIP_R_16F, ldb, beta,
                               (void *)d_C, HIP_R_32F, ldc);
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            if (stat != HIPBLAS_STATUS_SUCCESS)
            {
                cerr << "hipblasSgemm failed" << endl;
                exit(1);
            }
            assert(!hipGetLastError());

            float elapsed; // ms returned
            hipEventElapsedTime(&elapsed, start, stop);
            elapsed /= 1000.0f;
            sum += elapsed; // seconds
            totalSum += sum;
        }
        long ops = 2 * (long)m * (long)n * (long)k;
        double opss = (double)ops / (double)(sum / repeats) / (double)1000000000000;
        cout << fixed << setprecision(4) << "float_Mix: m, n, k = ["        
            << size << "]" << ", Average: " << sum/repeats << " secs, "
            << "TFlops [" << opss << "], Total:" << totalSum/repeats << " secs" << endl;

    }


    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free CPU memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;

}
